#include "hip/hip_runtime.h"
#include "rw_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

__global__ void uniform_random_walk_kernel(const int64_t *rowptr,
                                           const int64_t *col,
                                           const int64_t *start,
                                           const float *rand, int64_t *out,
                                           int64_t walk_length, int64_t numel) {
  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel) {
    out[thread_idx] = start[thread_idx];

    int64_t row_start, row_end, i, cur;
    for (int64_t l = 1; l <= walk_length; l++) {
      i = (l - 1) * numel + thread_idx;
      cur = out[i];
      row_start = rowptr[cur], row_end = rowptr[cur + 1];

      out[l * numel + thread_idx] =
          col[row_start + int64_t(rand[i] * (row_end - row_start))];
    }
  }
}

torch::Tensor random_walk_cuda(torch::Tensor rowptr, torch::Tensor col,
                               torch::Tensor start, int64_t walk_length,
                               double p, double q) {
  CHECK_CUDA(rowptr);
  CHECK_CUDA(col);
  CHECK_CUDA(start);
  hipSetDevice(rowptr.get_device());

  CHECK_INPUT(rowptr.dim() == 1);
  CHECK_INPUT(col.dim() == 1);
  CHECK_INPUT(start.dim() == 1);

  auto rand = torch::rand({start.size(0), walk_length},
                          start.options().dtype(torch::kFloat));
  auto out = torch::full({walk_length + 1, start.size(0)}, -1, start.options());

  auto stream = at::cuda::getCurrentCUDAStream();
  uniform_random_walk_kernel<<<BLOCKS(start.numel()), THREADS, 0, stream>>>(
      rowptr.data_ptr<int64_t>(), col.data_ptr<int64_t>(),
      start.data_ptr<int64_t>(), rand.data_ptr<float>(),
      out.data_ptr<int64_t>(), walk_length, start.numel());

  return out.t().contiguous();
}
